#ifndef _CUDA_UTILS_CU_
#define _CUDA_UTILS_CU_

void 
cudaSucceed(const char *msg)
{
  hipError_t err;
  hipDeviceSynchronize ();
  err = hipGetLastError();
    
  if (err == hipSuccess)
  {
    return; 
  }
  else
  {
    fprintf (stderr, "CUDA error: %s: %s.\n", msg, hipGetErrorString (err));
    exit(EXIT_FAILURE);
  }
}

#endif
